
#include <hip/hip_runtime.h>


__global__ void MatrixAdd(float *Mat1,float *Mat2,float *Mat3, int n, int p){

    int idx= blockIdx.x* blockDim.x+ threadIdx.x;

    if (idx<n*p){
        Mat3[idx]=Mat2[idx]+Mat1[idx];
    }
}
