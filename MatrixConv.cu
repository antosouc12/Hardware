#include "hip/hip_runtime.h"
#include "activation.cu"


__global__ void convolve2d(float* C, float* M, float* K, int n, int ksize){

    int idx= blockDim.x * blockIdx.x + threadIdx.x;
    int x= (idx%(n*n))/n;  // La variable x nous permet de se deplacer selon les lignes 
    int y= idx%n;  //La variable y nous permet de se deplacer selon les colonnes 
    int w = idx/(n*n);  //La variable w nous permet de se deplacer selon la profondeur
    float tmp = float(0);
    
    for(int i=0; i<ksize;i++){
        for (int j=0; j<ksize;j++){
            tmp+=K[(ksize-i-1)*ksize+(ksize-j-1)+w*5*5]*M[(x+i)*n+y+j+w*n*n];
        }
    }
    
    C[idx]=activation(tmp/(n*n));   // Nous passons la valeur dans une fonction d'activation qui est ici la fonction tanh
}


