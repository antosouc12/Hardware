#include "hip/hip_runtime.h"
#include "activation.cu"


__global__ void convolve2d(float* C, float* M, float* K, int n, int ksize){

    int idx= blockDim.x * blockIdx.x + threadIdx.x;
    int x= (idx%(n*n))/n;
    int y= idx%n; 
    int w = idx/(n*n);
    float tmp = float(0);
    
    for(int i=0; i<ksize;i++){
        for (int j=0; j<ksize;j++){
            tmp+=K[(ksize-i-1)*ksize+(ksize-j-1)+w*5*5]*M[(x+i)*n+y+j];
        }
    }
    
    C[idx]=activation(tmp/(n*n));
}


