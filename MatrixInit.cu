
#include <hip/hip_runtime.h>
#include <hiprand/hiprand_kernel.h>


__global__ void FillMatrix(float* M, int n){

    int idx=blockIdx.x*blockDim.x+threadIdx.x;
    hiprandState state;
    hiprand_init((unsigned long long)clock() + idx, 0, 0, &state);
    
    if (idx<n){
        M[idx]=hiprand_uniform_double(&state);
    }
    
}


