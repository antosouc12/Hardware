
#include <hip/hip_runtime.h>
#include <hiprand/hiprand_kernel.h> //Curand kernel est une library qui nous permet de generer des valeurs aleatoires a l'interieur du GPU


__global__ void FillMatrix(float* M, int n){

    int idx=blockIdx.x*blockDim.x+threadIdx.x;
    hiprandState state;
    hiprand_init((unsigned long long)clock() + idx, 0, 0, &state);       //Ces deux lignes permettent d'initer le generateur aleatoire
    
    if (idx<n){
        M[idx]=hiprand_uniform_double(&state);
    }
    
}


