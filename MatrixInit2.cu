

#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <hiprand/hiprand_kernel.h>

__global__ void MatrixCoefPrint(float** M, int n){

    int idx= blockIdx.x * blockDim.x + threadIdx.x;
    int i = idx/n;
    int j = idx%n;

    if (j==(n-1)){
        printf("%f \n" , M[i][j]);
    }
    else{
        printf("%f", M[i][j]);
    }
}


__host__ int main(void){

    int n=5;
    int p=5;

    float **M_h, **M_d;

    M_h=(float**)malloc(n*sizeof(float*));
    for (int m=0;m<n;m++){
        M_h[m]=(float*)malloc(p*sizeof(float));
    }

    int Nthread=1024;
    int Nblock= (n*p+Nthread)/Nthread;

    for (int i=0;i<n;i++){
        for(int j=0;j<p;j++){
            M_h[i][j]=(float)i+j;
        }
    }

    hipMalloc((void **) &M_d, n*sizeof(float*));

    for(int i=0;i<n;i++){
        hipMalloc((void **) &(M_d[i]), p*sizeof(float));
        hipMemcpy(M_d[i],M_h[i],p*sizeof(float),hipMemcpyHostToDevice);
    }
    
    MatrixCoefPrint<<<Nblock,Nthread>>>(M_d,n);

    for(int i=0;i<n;i++){
        hipMemcpy(M_h[i],M_d[i],p*sizeof(float),hipMemcpyDeviceToHost);
    }





    

}

