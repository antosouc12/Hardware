
#include <hip/hip_runtime.h>
// #include "MatrixPrint.cu"

__global__ void MatMoy(float* M_o,float* M_i, int n){

    int idx=blockIdx.x*blockDim.x+threadIdx.x;

    int m=n/2;
    int x=(idx%(m*m))/m;
    int y=idx%(m);
    int w=idx/(m*m);

    M_o[idx]= (M_i[2*(x*n+y)+w*n*n]+M_i[2*(x*n+y)+1+w*n*n]+M_i[(2*x+1)*n+2*y+w*n*n]+M_i[(2*x+1)*n+2*y+1+w*n*n])/4;

    if(idx==4){
        printf("M_i[2*(x*n+y)+w*n*n] = %f \n", M_i[2*(x*n+y)+w*n*n]);
        printf("M_i[2*(x*n+y)+1+w*n*n] = %f \n", M_i[2*(x*n+y)+1+w*n*n]);
        printf("M_i[2*(x*n+y)+n+w*n*n] = %f \n", M_i[2*(x*n+y)+n+w*n*n]);
        printf("M_i[2*(x*n+y)+n+1+w*n*n] = %f \n", M_i[2*(x*n+y)+n+1+w*n*n]);
        printf("x = %d \n", x);
        printf("y = %d \n", y);
    }
}


// __host__ int main(){

//     int n=8;
//     int p=8;
//     int h=6;

//     float * M_h1, *M_h2, *M_d1, *M_d2;

//     M_h1=(float*)malloc(n*p*h*sizeof(float));
//     M_h2=(float*)malloc(n*p*h*sizeof(float)/4);

//     cudaMalloc((void **) &M_d1, n*p*h*sizeof(float));
//     cudaMalloc((void **) &M_d2, n*p*h*sizeof(float)/4);

//     for (int i=0;i<n*p*h;i++){
//         M_h1[i]=(float) i;
//     }

//     MatrixPrint(M_h1,n,p);
//     MatrixPrint(M_h2,n/2,p/2);

//     cudaMemcpy(M_d1,M_h1,n*p*h*sizeof(float),cudaMemcpyHostToDevice);

//     int Nthread= p*h;
//     int Nblock = n;

//     MatMoy<<<Nblock,Nthread>>>(M_d2,M_d1,8);

//     cudaMemcpy(M_h2,M_d2,n*p*h*sizeof(float)/4,cudaMemcpyDeviceToHost);

//     MatrixPrint(M_h2,n/2,p/2);
// }

