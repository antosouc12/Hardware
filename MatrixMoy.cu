
#include <hip/hip_runtime.h>

__global__ void MatMoy(float* M_o,float* M_i, int n){

    int idx=blockIdx.x*blockDim.x+threadIdx.x;

    int m=n/2;
    int x=(idx%(m*m))/m;
    int y=idx%(m);
    int w=idx/(m*m);

    M_o[idx]= (M_i[2*(x*n+y)+w*n*n]+M_i[2*(x*n+y)+1+w*n*n]+M_i[(2*x+1)*n+2*y+w*n*n]+M_i[(2*x+1)*n+2*y+1+w*n*n])/4;

}
