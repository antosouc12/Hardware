
#include <hip/hip_runtime.h>


__global__ void MatrixMul(float *M1,float *M2,float *M3, int n){

    int idx= blockIdx.x * blockDim.x + threadIdx.x;

    int l=idx/n;
    int k=idx%n;
    float tmp;
    for (int i=0; i<n;i++){
        tmp+=M1[l+i*n]*M2[i*n+k];
    }
    M3[idx]=tmp;

}
