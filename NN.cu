#include <stdio.h>
#include <stdlib.h>
#include <hiprand/hiprand_kernel.h>
#include "MatrixInit.cu"
#include "MatrixConv.cu"
#include "MatrixMoy.cu"
#include "MatrixPrint.cu"

__host__ int main(){    


    int n=32;
    int p=32;
    int size_f=sizeof(float);
    int Nthread=1024;
    int Nblock=(n*p+Nthread)/Nthread;

    float *raw_data;
    raw_data=(float*)malloc(n*p*sizeof(float));
    float *M_d;
    printf("You are at hipMalloc \n");
    hipMalloc((void **) &M_d,size_f*n*p);
    printf("You are at FillMatrix \n");
    FillMatrix<<<Nblock,Nthread>>>(M_d,n*p);

    hipMemcpy(raw_data,M_d,n*p*size_f,hipMemcpyDeviceToHost);

    MatrixPrint(raw_data,32,32);
    
    float *C1_kernel;
    C1_kernel=(float *)malloc(size_f*5*5*6);

    printf("You are at for loop \n");
    for(int i=0;i<6;i++){
        printf("You are at FillMatrix 2 \n");
        FillMatrix<<<Nblock,Nthread>>>(M_d,5*5);
        hipMemcpy(C1_kernel+i*5*5,M_d,5*5*size_f,hipMemcpyDeviceToHost);
    }

    MatrixPrint(C1_kernel,5,5);

    // float C1_data[6*28*28];
    // float S1_data[6*14*14];


    float *C1_data;
    float *S1_data;
    C1_data=(float*)malloc(28*28*6*sizeof(float));
    S1_data=(float*)malloc(14*14*6*sizeof(float));
    
    float *C1_data_d;
    float *C1_kernel_d;
    float *S1_data_d;
    hipMalloc((void **) &C1_kernel_d, 5*5*6*sizeof(float));
    hipMalloc((void **) &C1_data_d, 28*28*6*sizeof(float));
    hipMalloc((void **) &S1_data_d, 14*14*6*sizeof(float));

    hipMemcpy(C1_kernel_d,C1_kernel, 5*5*6*sizeof(float),hipMemcpyHostToDevice);
    hipMemcpy(M_d,raw_data,32*32*6*sizeof(float),hipMemcpyHostToDevice);


    // for(int i=0;i<28*28*6;i++){
    //     C1_data[i]=(float)i;
    // }

    // for (int i=0;i<14*14*6;i++){
    //     S1_data[i]=(float)i;
    // }
    // printf("You are at memset \n");

    // memset(C1_data, 5, sizeof(C1_data));
    // memset(S1_data, 5, sizeof(S1_data));

    // MatrixPrint(C1_data,28,28);
    // MatrixPrint(S1_data,14,14);

    // hipMemcpy(C1_data_d,C1_data,28*28*6*size_f,hipMemcpyHostToDevice);
    // hipMemcpy(S1_data_d,S1_data,14*14*6*size_f,hipMemcpyHostToDevice);

    Nblock=28;
    Nthread=28*6;

    //Premiere couche

    // printf("You are at for loop 2 \n");
    // for(int i=0;i<6;i++){
    //     printf("You are at convolve 1\n");
    //     convolve2d<<<Nblock,Nthread>>>(C1_data_d+i*28*28,M_d,C1_kernel_d+i*5*5,32,5);
    //     hipDeviceSynchronize();
    //     printf("You are at convolve 2\n");
    // }
    
    convolve2d<<<Nblock,Nthread>>>(C1_data_d,M_d,C1_kernel_d,32,5);

    hipMemcpy(C1_data,C1_data_d,28*28*6*size_f,hipMemcpyDeviceToHost);
    //hipFree(C1_data_d);
    
    MatrixPrint(C1_data,28,28);
    
    //Seconde couche 
    
    printf("You are at matmoy \n");

    MatMoy<<<Nblock,Nthread>>>(S1_data_d,C1_data_d,28);

    hipMemcpy(S1_data,S1_data_d,14*14*6,hipMemcpyDeviceToHost);

    MatrixPrint(S1_data,14,14);

    printf("You are done \n");

  

    
}