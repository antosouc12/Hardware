#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <hiprand/hiprand_kernel.h>
#include "MatrixInit.cu"
#include "MatrixConv.cu"
#include "MatrixMoy.cu"
#include "MatrixPrint.cu"

__host__ int main(){    


    int n=32;
    int p=32;
    int size_f=sizeof(float);
    int Nthread=1024;
    int Nblock=(n*p+Nthread)/Nthread;
`   
    float *raw_data;    // C'est la matrice dans laquelle se trouvera notre image d'origine dans le CPU
    raw_data=(float*)malloc(n*p*sizeof(float));
    float *M_d;     // C'est la matrice dans laquelle se trouvera notre image d'origine dans le GPU
    hipMalloc((void **) &M_d,size_f*n*p);
    FillMatrix<<<Nblock,Nthread>>>(M_d,n*p); // On remplit la matrice dans le GPU

    hipMemcpy(raw_data,M_d,n*p*size_f,hipMemcpyDeviceToHost); 

    MatrixPrint(raw_data,32,32);
    
    float *C1_kernel;   //C'est la matrice dans laquelle nos 6 kernels se trouveront dans le CPU
    C1_kernel=(float *)malloc(size_f*5*5*6);

    for(int i=0;i<6;i++){
        FillMatrix<<<Nblock,Nthread>>>(M_d,5*5);  //Nous remplissons la matrice dans le GPU cependant nous faisons une boucle dans le CPU 6 fois pour la remplir.
                                                  // Nous aurions pu faire tout directement dans le GPU mais cela facilite le code 
        hipMemcpy(C1_kernel+i*5*5,M_d,5*5*size_f,hipMemcpyDeviceToHost);
    }

    MatrixPrint(C1_kernel,5,5);

    float *C1_data; // C'est la matrice dans laquelle se trouvent les valeurs apres la premiere couche de convolution dans le CPU
    float *S1_data; // C'est la matrice dans laquelle se trouvent les valeurs apres la premiere couche de moyenne pooling dans le CPU
    C1_data=(float*)malloc(28*28*6*sizeof(float));
    S1_data=(float*)malloc(14*14*6*sizeof(float));
    
    float *C1_data_d;  // C'est la matrice dans laquelle se trouvent les valeurs apres la premiere couche de convolution dans le GPU
    float *C1_kernel_d;  //C'est la matrice dans laquelle nos 6 kernels se trouveront dans le GPU
    float *S1_data_d;  // C'est la matrice dans laquelle se trouvent les valeurs apres la premiere couche de moyenne pooling dans le GPU
    hipMalloc((void **) &C1_kernel_d, 5*5*6*sizeof(float));
    hipMalloc((void **) &C1_data_d, 28*28*6*sizeof(float));
    hipMalloc((void **) &S1_data_d, 14*14*6*sizeof(float));

    hipMemcpy(C1_kernel_d,C1_kernel, 5*5*6*sizeof(float),hipMemcpyHostToDevice);
    hipMemcpy(M_d,raw_data,32*32*6*sizeof(float),hipMemcpyHostToDevice);

    Nblock=28;
    Nthread=28*6;

    //Premiere couche
    
    convolve2d<<<Nblock,Nthread>>>(C1_data_d,M_d,C1_kernel_d,32,5); //On fait attention de bien utiliser les matrices GPUs lorsque l'on appelle les fonctions __global__ ou __device__

    hipMemcpy(C1_data,C1_data_d,28*28*6*size_f,hipMemcpyDeviceToHost); //On copie bien l'information des matrices GPU vers les matrices CPU
    
    MatrixPrint(C1_data,28,28); //On fait attention de bien utiliser les matrices GPUs lorsque l'on appelle les fonctions __host__
    
    //Seconde couche 

    MatMoy<<<Nblock,Nthread>>>(S1_data_d,C1_data_d,28);

    hipMemcpy(S1_data,S1_data_d,14*14*6,hipMemcpyDeviceToHost);

    MatrixPrint(S1_data,14,14);

}
