#include <stdio.h>
#include <stdlib.h>
#include <hiprand/hiprand_kernel.h>
#include "MatrixInit.cu"
#include "MatrixConv.cu"
#include "MatrixMoy.cu"
#include "MatrixPrint.cu"

__host__ int main(){    


    int n=32;
    int p=32;
    int size_f=sizeof(float);
    int Nthread=1024;
    int Nblock=(n*p+Nthread)/Nthread;

    float *raw_data;
    raw_data=(float*)malloc(n*p*sizeof(float));
    float *M_d;
    hipMalloc((void **) &M_d,size_f*n*p);
    FillMatrix<<<Nblock,Nthread>>>(M_d,n*p);

    hipMemcpy(raw_data,M_d,n*p*size_f,hipMemcpyDeviceToHost);

    MatrixPrint(raw_data,32,32);
    
    float *C1_kernel;
    C1_kernel=(float *)malloc(size_f*5*5*6);

    for(int i=0;i<6;i++){
        FillMatrix<<<Nblock,Nthread>>>(M_d,5*5);
        hipMemcpy(C1_kernel+i*5*5,M_d,5*5*size_f,hipMemcpyDeviceToHost);
    }

    MatrixPrint(C1_kernel,5,5);

    float *C1_data;
    float *S1_data;
    C1_data=(float*)malloc(28*28*6*sizeof(float));
    S1_data=(float*)malloc(14*14*6*sizeof(float));
    
    float *C1_data_d;
    float *C1_kernel_d;
    float *S1_data_d;
    hipMalloc((void **) &C1_kernel_d, 5*5*6*sizeof(float));
    hipMalloc((void **) &C1_data_d, 28*28*6*sizeof(float));
    hipMalloc((void **) &S1_data_d, 14*14*6*sizeof(float));

    hipMemcpy(C1_kernel_d,C1_kernel, 5*5*6*sizeof(float),hipMemcpyHostToDevice);
    hipMemcpy(M_d,raw_data,32*32*6*sizeof(float),hipMemcpyHostToDevice);

    Nblock=28;
    Nthread=28*6;

    //Premiere couche
    
    convolve2d<<<Nblock,Nthread>>>(C1_data_d,M_d,C1_kernel_d,32,5);

    hipMemcpy(C1_data,C1_data_d,28*28*6*size_f,hipMemcpyDeviceToHost);
    
    MatrixPrint(C1_data,28,28);
    
    //Seconde couche 

    MatMoy<<<Nblock,Nthread>>>(S1_data_d,C1_data_d,28);

    hipMemcpy(S1_data,S1_data_d,14*14*6,hipMemcpyDeviceToHost);

    MatrixPrint(S1_data,14,14);

}
