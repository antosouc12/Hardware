
#include <hip/hip_runtime.h>





__global__ void activation(float* M_o, float* M_i, int n){

    int idx= blockDim.x*blockIdx.x + threadIdx.x;

    M_o[idx]=tanh(M_i[idx]);
}