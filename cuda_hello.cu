
#include <hip/hip_runtime.h>
#include <stdio.h> 

__global__ void cuda_hello(){
    printf("Hello World, from GPU! \n");
    printf("You are in the %d Block \n", blockIdx.x);
    printf("You are in the %d thread \n", threadIdx.x);

    
}

int main(){
    //__global__ printf("The dimensions of your block is %d", blockDim.x); 
    cuda_hello<<<2,6>>>();

    hipDeviceSynchronize();
    return 0;
}