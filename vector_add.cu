
#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void vector_add(float* a, float* b, float* c){

    int idx=blockIdx.x *blockDim.x + threadIdx.x;
    
    c[idx]=a[idx]+b[idx];
    //printf("%d \n", threadIdx.x);
    //printf("%d \n", blockIdx.x);


}

__host__ int main(void){

    int N=10000000;
    int size= sizeof(float);
    float *a_h,*b_h,*c_h,*a_d,*b_d,*c_d ;
    int Nthread=1024;
    int Nblock= (N+Nthread)/Nthread;

    a_h=(float*)malloc(N*size);
    b_h=(float*)malloc(N*size);
    c_h=(float*)malloc(N*size);

    for (int i=0;i<N;i++){
        a_h[i]=(float) 3;
        b_h[i]=(float) 5;
    }

    hipMalloc((void **) &a_d, N*size);
    hipMalloc((void **) &b_d, N*size);
    hipMalloc((void **) &c_d, N*size);

    hipMemcpy(a_d,a_h,N*size,hipMemcpyHostToDevice);
    hipMemcpy(b_d,b_h,N*size,hipMemcpyHostToDevice);
    hipMemcpy(c_d,c_h,N*size,hipMemcpyHostToDevice);

    vector_add<<<Nblock,Nthread>>>(a_d,b_d,c_d);

    hipMemcpy(a_h,a_d,N*size,hipMemcpyDeviceToHost);
    hipMemcpy(b_h,b_d,N*size,hipMemcpyDeviceToHost);
    hipMemcpy(c_h,c_d,N*size,hipMemcpyDeviceToHost);

    // cudaFree(a_d);
    // cudaFree(b_d);
    // cudaFree(c_d);

    // free(a_h);
    // free(b_h);
    // free(c_h);

    printf("%f \n", c_h[0]);
    
}